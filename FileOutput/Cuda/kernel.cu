﻿#include "hip/hip_runtime.h"				// for funcs like cudaMalloc and defines like __global__
	// for paramteres like threadIdx

constexpr int WIDTH = 640;		// number of pixels on x-axis;
constexpr int HEIGHT = 360;		// number of pixels on y-axis;
constexpr int DEPTH = 24;		// color depth in bits;

#define SIZE		HEIGHT * WIDTH	// total image pixels
#define BITS		SIZE * DEPTH	// total image size in bits
#define BYTES		BITS / 8		// total image size in bytes
#define CHANNELS	DEPTH / 8		// number of bytes per color

typedef char color[CHANNELS];

__device__ void shader(color& out, int x, int y)
{
	// calculate color (same for each channel)
	char color = 255 / 3 * 2;

	// apply same color on each channel
	for (int i = 0; i < CHANNELS; ++i)
	{
		out[i] = color;
	}
}

__global__ void applyShader(color* data)
{
	int i = threadIdx.x;

	shader(data[i], i, i);
}

int main()
{
	color* cudaData = nullptr;

	hipMalloc(&cudaData, BYTES);
	
	applyShader<<<1, BYTES>>>(cudaData);

	color* outData = new color[BYTES];

	hipMemcpy(outData, cudaData, BYTES, hipMemcpyDeviceToHost);
}
